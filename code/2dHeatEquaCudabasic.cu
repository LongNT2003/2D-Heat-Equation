// %%writefile 2dHeatEquacuda.cu
// this implement with only 1 dimension grid and block
#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
#define N 1024
#define c 0.002
#define delta_t 0.05
#define delta_s 0.04
#define Ntime 1000
#define GridSize 16
#define BlockSize 16
#define ThreadSize N/(GridSize*BlockSize)
void initData(float *T){
    for (int i=0;i<N;i++){
        for (int j=0;j<N;j++){
            *(T+i*(N)+j)=25.0;
        }
    }

}
void printData(float *T) {
    for (int i = 0; i <= N -1; i++) {
        for (int j = 0; j <= N -1; j++) {
            printf("%6.1f ", *(T + i * (N) + j));
        }
        printf("\n");
    }
}
//=========================
__global__ void Derivative(float *T, float *dT){
    float up,down,left,right,cen;
	int row, col, index, start, stop;
	index = blockIdx.x * blockDim.x + threadIdx.x;
    start = index*ThreadSize;
    stop  = start + ThreadSize;         
    for (row=start;row<stop;row++) {
        for (col=0;col<N;col++){
            cen=*(T+row*N+col);
            up   = (row==0) ? 25.0 : *(T+ (row-1)*(N) +col);
            down = (row==N-1) ? 25.0 : *(T+ (row+1)*(N) +col);
            right= (col==N-1) ? 100.0 : *(T+ row*(N) +col+1);
            left = (col==0) ? 25.0 : *(T+ row*(N) +col-1);
            *(dT+row*(N)+col) = c*(up+down+left+right-4*cen)/(delta_s*delta_s);
        }
      
    }
    __syncthreads();
}
__global__ void SolvingODE(float *T,float *dT) 
{
	int row, col, index, start, stop;
	index = blockIdx.x * blockDim.x + threadIdx.x;
    start = index*ThreadSize;
    stop  = start + ThreadSize;         
    for (row=start;row<stop;row++) {
        for (col=0;col<N;col++){
            *(T+row*(N)+col) = *(T+row*(N)+col) + delta_t*(*(dT+row*(N)+col));
        }
    }
	__syncthreads();
}
//=========================
int main(int argc, char **argv){
    //1a. Delare and Allocate Mem on CPU
    float *Tcpu,*dTcpu;
    Tcpu=(float *)malloc((N) * (N) * sizeof(float));
    dTcpu=(float *)malloc((N) * (N) * sizeof(float));
    initData(Tcpu);
    //1b. Delare and Allocate Mem on GPU
    float *Tgpu,*dTgpu;
    hipMalloc((void**)&Tgpu ,N*N*sizeof(int));
    hipMalloc((void**)&dTgpu,N*N*sizeof(int));
    //2. Copy Input from CPU to GPU
    hipMemcpy(Tgpu,Tcpu,N*N*sizeof(int),hipMemcpyHostToDevice);
    //3. Define Block and Thread Structure
    dim3 dimGrid(GridSize);
    dim3 dimBlock(BlockSize);
    for (int t=0;t<Ntime;t++) {
             Derivative<<<dimGrid,dimBlock>>>(Tgpu,dTgpu);
             SolvingODE<<<dimGrid,dimBlock>>>(Tgpu,dTgpu);
         }
    //5. Copy Output from GPU to CPU
    hipMemcpy(Tcpu,Tgpu,N*N*sizeof(int),hipMemcpyDeviceToHost);
    printData(Tcpu);
    //6. Free Mem on CPU and GPU
    free(Tcpu);free(dTcpu);
    hipFree(Tgpu);hipFree(dTgpu);
    return 0;
}